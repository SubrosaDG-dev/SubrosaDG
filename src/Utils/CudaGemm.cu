#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm_batched.h>

#include <Eigen/Core>
#include <cuda/api.hpp>

#include "Utils/CudaGemm.cuh"

cuda::device_t device = cuda::device::current::get();

cutlass::gemm::device::GemmBatched<real, cutlass::layout::ColumnMajor, real, cutlass::layout::ColumnMajor, real,
                                   cutlass::layout::ColumnMajor>
    gemm;

Matrix::Matrix() {
  Eigen::Matrix<real, kN, Eigen::Dynamic> h_a(kN, kBatchSize * kN);
  Eigen::Matrix<real, kN, kN> h_b;

  h_a.setRandom();
  h_b.setRandom();

  cuda::memory::copy(d_a_, h_a.data(), kBatchSize * kN * kN * sizeof(real));
  cuda::memory::copy(d_b_, h_b.data(), kN * kN * sizeof(real));
}

float cudaComputation(Matrix& matrix) {
  std::pair<cuda::event_t, cuda::event_t> events = std::make_pair(device.create_event(), device.create_event());
  cuda::stream_t stream = device.default_stream();

  stream.enqueue.event(events.first);
  gemm({{kN, kN, kN},
        {matrix.d_a_.data(), kN},
        kN * kN,
        {matrix.d_b_.data(), kN},
        0,
        {matrix.d_c_.data(), kN},
        kN * kN,
        {matrix.d_c_.data(), kN},
        kN * kN,
        {1.0, 0.0},
        kBatchSize});
  stream.enqueue.event(events.second);
  stream.synchronize();

  return cuda::event::time_elapsed_between(events).count();
}
