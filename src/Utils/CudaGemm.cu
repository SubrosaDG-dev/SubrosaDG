#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <Eigen/Core>
#include <cuda/api.hpp>

#include "Utils/CudaGemm.cuh"

__global__ void gemm(const real* d_a, const real* d_b, real* d_c) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < kBatchSize) {
    Eigen::Map<const Eigen::Matrix<real, 4, kN>> a(d_a + i * 4 * kN);
    Eigen::Map<const Eigen::Matrix<real, kN, kN>> b(d_b);
    Eigen::Map<Eigen::Matrix<real, 4, kN>> c(d_c + i * 4 * kN);
    a.row(0) * b.col(0);
  }
}

Matrix::Matrix() {
  Eigen::Array<Eigen::Matrix<real, 4, kN>, Eigen::Dynamic, 1> h_a(kBatchSize);
  Eigen::Matrix<real, kN, kN> h_b;

  for (int i = 0; i < kBatchSize; i++) {
    h_a[i].setRandom();
  }
  h_b.setRandom();

  cuda::memory::copy(d_a, h_a.data(), kBatchSize * 4 * kN * sizeof(real));
  cuda::memory::copy(d_b, h_b.data(), kN * kN * sizeof(real));
}

float cudaComputation(Matrix& matrix) {
  cuda::device_t device = cuda::device::current::get();

  cuda::launch_configuration_t launch_config =
      cuda::launch_config_builder().overall_size(kBatchSize).block_size(kThreadsPerBlock).build();

  std::pair<cuda::event_t, cuda::event_t> events = std::make_pair(device.create_event(), device.create_event());
  cuda::stream_t stream = device.default_stream();

  cuda::launch(cudaEigenComputation, launch_config, d_a.data(), d_b.data(), d_c.data());

  stream.enqueue.event(events.first);

  for (int i = 0; i < 5; i++) {
    cuda::launch(cudaEigenComputation, launch_config, d_a.data(), d_b.data(), d_c.data());
  }

  stream.enqueue.event(events.second);
  stream.synchronize();

  return cuda::event::time_elapsed_between(events).count();
}
